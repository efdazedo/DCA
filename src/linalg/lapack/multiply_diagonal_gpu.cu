#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Raffaele Solca' (rasolca@itp.phys.ethz.ch)
//
// This file implements laset_gpu.hpp.

#include "dca/linalg/lapack/multiply_diagonal_gpu.hpp"
#include <cassert>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/linalg/util/stream_functions.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace linalg {
namespace lapack {
namespace kernels {
// dca::linalg::lapack::kernels::

constexpr int multiply_diag_block_size_x = 128;
constexpr int multiply_diag_block_size_y = 32;

template <typename ScalarIn, typename ScalarOut>
__global__ void multiplyDiagonalLeft(int m, int n, const ScalarIn* d, int inc_d, const ScalarIn* a,
                                     int lda, ScalarOut* b, int ldb) {
  // Work on a tile of size (blockDim.x x multiply_diag_block_size_y).
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < m) {
    int js = blockIdx.y * multiply_diag_block_size_y;
    int je = min(n, (blockIdx.y + 1) * blockDim.x);

    for (int j = js; j < je; ++j)
      b[i + j * ldb] = d[i * inc_d] * a[i + j * lda];
  }
}

template <typename Type>
__global__ void multiplyDiagonalRight(int m, int n, const Type* a, int lda, const Type* d,
                                      int inc_d, Type* b, int ldb) {
  // Work on a tile of size (blockDim.x x multiply_diag_block_size_y).
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < m) {
    int js = blockIdx.y * multiply_diag_block_size_y;
    int je = min(n, (blockIdx.y + 1) * blockDim.x);

    for (int j = js; j < je; ++j)
      b[i + j * ldb] = d[j * inc_d] * a[i + j * lda];
  }
}

}  // kernels
// dca::linalg::lapack::

template <typename ScalarIn, typename ScalarOut>
void multiplyDiagonalLeft_gpu(int m, int n, const ScalarIn* d, int inc_d, const ScalarIn* a,
                              int lda, ScalarOut* b, int ldb, int thread_id, int stream_id) {
  assert(lda >= m);
  assert(ldb >= m);

  if (m > 0 && n > 0) {
    checkErrorsCudaDebug();
    int bl_x = dca::util::ceilDiv(m, kernels::multiply_diag_block_size_x);
    int bl_y = dca::util::ceilDiv(n, kernels::multiply_diag_block_size_y);

    dim3 threads(kernels::multiply_diag_block_size_x);
    dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::multiplyDiagonalLeft<ScalarIn, ScalarOut>
        <<<blocks, threads, 0, stream>>>(m, n, d, inc_d, a, lda, b, ldb);
    checkErrorsCudaDebug();
  }
}

template void multiplyDiagonalLeft_gpu<float, float>(int m, int n, const float* d, int inc_d,
                                                     const float* a, int lda, float* b, int ldb,
                                                     int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<double, double>(int m, int n, const double* d, int inc_d,
                                                       const double* a, int lda, double* b, int ldb,
                                                       int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<double, float>(int m, int n, const double* d, int inc_d,
                                                      const double* a, int lda, float* b, int ldb,
                                                      int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<hipComplex, hipComplex>(int m, int n, const hipComplex* d,
                                                             int inc_d, const hipComplex* a, int lda,
                                                             hipComplex* b, int ldb, int thread_id,
                                                             int stream_id);
template void multiplyDiagonalLeft_gpu<hipDoubleComplex, hipDoubleComplex>(
    int m, int n, const hipDoubleComplex* d, int inc_d, const hipDoubleComplex* a, int lda,
    hipDoubleComplex* b, int ldb, int thread_id, int stream_id);

template <typename Type>
void multiplyDiagonalRight_gpu(int m, int n, const Type* a, int lda, const Type* d, int inc_d,
                               Type* b, int ldb, int thread_id, int stream_id) {
  assert(lda >= m);
  assert(ldb >= m);

  if (m > 0 && n > 0) {
    checkErrorsCudaDebug();
    int bl_x = dca::util::ceilDiv(m, kernels::multiply_diag_block_size_x);
    int bl_y = dca::util::ceilDiv(n, kernels::multiply_diag_block_size_y);

    dim3 threads(kernels::multiply_diag_block_size_x);
    dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::multiplyDiagonalRight<<<blocks, threads, 0, stream>>>(m, n, a, lda, d, inc_d, b, ldb);
    checkErrorsCudaDebug();
  }
}
template void multiplyDiagonalRight_gpu(int m, int n, const float* a, int lda, const float* d,
                                        int inc_d, float* b, int ldb, int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const double* a, int lda, const double* d,
                                        int inc_d, double* b, int ldb, int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const hipComplex* a, int lda,
                                        const hipComplex* d, int inc_d, hipComplex* b, int ldb,
                                        int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const hipDoubleComplex* a, int lda,
                                        const hipDoubleComplex* d, int inc_d, hipDoubleComplex* b,
                                        int ldb, int thread_id, int stream_id);

}  // lapack
}  // linalg
}  // dca
