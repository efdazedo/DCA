#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements tensorcore_gemm.hpp

#include "dca/linalg/blas/tensorcore_gemm.hpp"

#include <iostream>
#include <hipblas.h>

#include "dca/linalg/util/handle_functions.hpp"
#include "dca/linalg/util/stream_functions.hpp"
#include "dca/linalg/matrixop.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace linalg {
namespace blas {
namespace kernel {
// dca::linalg::blas::kernel::

// Inverse transform of fp32_in[:] = fp16_out1[:] / scale1 + fp16_out2[:] / scale2
// TODO: multiply here and divide later.
void __global__ split(const int rows, const int cols, const float* fp32_in, const int ld_in,
                      const float scale, const float scale2, __half* fp16_out1, __half* fp16_out2,
                      const int ld_out) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= rows || j >= cols)
    return;

  const float original = fp32_in[i + ld_in * j];
  const __half high = __float2half(original / scale);
  fp16_out1[i + ld_out * j] = high;

  const float diff = original - __half2float(high);
  const __half low = __float2half(diff * scale / scale2);
  fp16_out2[i + ld_out * j] = low;
}
}  // namespace kernel
// dca::linalg::blas::

void tensorcoreGemm(const float alpha, const Matrix<float, GPU>& a, const Matrix<float, GPU>& b,
                    std::array<Matrix<__half, GPU>, 4>& workspace, const float beta,
                    Matrix<float, GPU>& c, int thread_id, int stream_id) {
  assert(a.nrCols() == b.nrRows());
  assert(a.nrRows() == c.nrRows());
  assert(b.nrCols() == c.nrCols());

  const float scale1 = 1.;  // TODO: change
  const float scale2 = scale1 * std::pow(2., -11);

  const dim3 threads(16, 16);
  using dca::util::ceilDiv;
  auto stream = util::getStream(thread_id, stream_id);

  auto split = [&](const auto& m, auto& high, auto& low) {
    high.resizeNoCopy(m.size());
    low.resizeNoCopy(m.size());

    dim3 blocks(ceilDiv(m.nrRows(), int(threads.x)), ceilDiv(m.nrCols(), int(threads.y)));
    kernel::split<<<blocks, threads, 0, stream>>>(m.nrRows(), m.nrCols(), m.ptr(),
                                                  m.leadingDimension(), scale1, scale2, high.ptr(),
                                                  low.ptr(), high.leadingDimension());
  };

  auto& a_high = workspace[0];
  auto& a_low = workspace[1];
  auto& b_high = workspace[2];
  auto& b_low = workspace[3];

  split(a, a_high, a_low);
  split(b, b_high, b_low);

  auto handle = util::getHandle(thread_id, stream_id);
  const int m = c.nrRows();
  const int n = c.nrCols();
  const int k = a.nrCols();

  auto multiply = [&](float alpha, const auto& a, const auto& b, float beta, auto& c) {
    auto err = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, a.ptr(), HIP_R_16F,
                            a.leadingDimension(), b.ptr(), HIP_R_16F, b.leadingDimension(), &beta,
                            c.ptr(), HIP_R_32F, c.leadingDimension(), HIP_R_32F,
                            CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    assert(err == HIPBLAS_STATUS_SUCCESS);
  };

  // c <- beta* c + alpha * (a_high * b_high) * scale1**2
  const auto alpha_11 = alpha * scale1 * scale1;
  multiply(alpha_11, a_high, b_high, beta, c);

  // c += alpha * (a_high * b_low) * scale1 * scale2
  const auto alpha_12 = alpha * scale2 * scale1;
  multiply(alpha_12, a_high, b_low, 1., c);

  // c += alpha * (a_low * b_high) * scale1 * scale2
  multiply(alpha_12, a_low, b_high, 1., c);
}

}  // namespace blas
}  // namespace linalg
}  // namespace dca
